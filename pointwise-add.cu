#include "hip/hip_runtime.h"
//  nvcc -O3 -std=c++11 -use_fast_math -ccbin g++ -arch=compute_75 -code=sm_75 -expt-relaxed-constexpr

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_fp16.h>
#include <mma.h>
#include <assert.h>
#include "common.h"

#define M 1024 
#define N 1024 
#define NUM_THREADS_PER_BLOCK 1024
#define vec 4
using namespace std;

__host__ void init_host_matrices(float *c){
  for (int t = 0; t < M * N; t++) {
    c[t] = (float) 0.0f;
  }
}

__host__ void printMatrixFloat(float* matrix, int m, int n){
  for(int i = 0; i < m; ++i){
    for(int j = 0; j < n; ++j){
      printf("%f ", (float)matrix[i * n + j]);
    }
    printf("\n");
  }
  printf("\n");
}

__global__ void pwAdd(float *c, int m, int n){
  float cst = 5;

  for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < m * n; i += (((m * n) + (NUM_THREADS_PER_BLOCK * vec) - 1) / (NUM_THREADS_PER_BLOCK * vec)) * blockDim.x * vec) {
    // Calculate this block's starting address.
    float *base = c + (i * vec);
    float4 *cGmem = (float4*)base;
    float4 cData = *(cGmem);

    cData.w = cData.w + cst;
    cData.x = cData.x + cst;
    cData.y = cData.y + cst;
    cData.z = cData.z + cst;

    *(cGmem) = cData;
    //printf("%f\n",(float)cData.w);
  }
}

int main() {
  float *d_c, *h_c, *h_c_gpu_res;
  int m, n;

  m = M;
  n = N;

  h_c = (float*) malloc(m * n * sizeof(float));
  h_c_gpu_res = (float*) malloc(m * n * sizeof(float));
  check_cuda_error(hipMalloc(&d_c, m * n * sizeof(float)));

  assert(((unsigned long long)d_c) % 128 == 0);

  init_host_matrices(h_c);
  check_cuda_error(hipMemcpy(d_c, h_c, m * n * sizeof(float), hipMemcpyHostToDevice));

  dim3 block(NUM_THREADS_PER_BLOCK, 1, 1);
  dim3 grid(((m * n) + (NUM_THREADS_PER_BLOCK * vec) - 1) / (NUM_THREADS_PER_BLOCK * vec), 1, 1);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, NULL);
  pwAdd<<<grid, block>>>(d_c, m , n);
  hipEventRecord(stop, NULL);

  hipEventSynchronize(stop);
  float msecTotal = 0.0f;
  hipEventElapsedTime(&msecTotal, start, stop);
  check_cuda_error(hipPeekAtLastError());
  //cout<<"time: "<<msecTotal<<"ms \n";

#ifdef PRINT_HOST
  check_cuda_error(hipDeviceSynchronize());
  hipMemcpy(h_c_gpu_res, d_c, m * n * sizeof(float), hipMemcpyDeviceToHost);
  check_cuda_error(hipDeviceSynchronize());
  printMatrixFloat(h_c_gpu_res, m, n);
#endif

  free(h_c);
  free(h_c_gpu_res);
  hipFree(d_c);

  return 0;
}
